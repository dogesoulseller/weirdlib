#include "hip/hip_runtime.h"
#include "../../include/cuda/weirdlib_cuda_image.hpp"
#include "./cuda_utils.cuh"
#ifdef WLIB_ENABLE_CUDA
#include <hip/hip_runtime.h>

namespace wlib
{
namespace image
{
namespace cu
{
	__global__ void kernel_NegateChannelValues(float* __restrict__ inout) {
		const int threadID = getGlobalIdx_2x1();
		inout[threadID] = 255.0f - inout[threadID];
	}

	void NegateValues(ImageSoACUDA& in, bool withAlpha) {
		switch (in.format)
		{
		case F_BGR:
		case F_Grayscale:
		case F_RGB:
		{
			for (size_t i = 0; i < in.channels.size(); i++) {
				hipStream_t stream;
				hipStreamCreate(&stream);
				kernel_NegateChannelValues<<<dim3(in.width, in.height), 1, 0, stream>>>(in.channels[i]);
				hipStreamDestroy(stream);
			}
		}
			break;
		case F_GrayAlpha:
		case F_BGRA:
		case F_RGBA: {
			if (!withAlpha) {
				for (size_t i = 0; i < in.channels.size()-1; i++) {
					hipStream_t stream;
					hipStreamCreate(&stream);
					kernel_NegateChannelValues<<<dim3(in.width, in.height), 1, 0, stream>>>(in.channels[i]);
					hipStreamDestroy(stream);
				}
			} else {
				for (size_t i = 0; i < in.channels.size(); i++) {
					hipStream_t stream;
					hipStreamCreate(&stream);
					kernel_NegateChannelValues<<<dim3(in.width, in.height), 1, 0, stream>>>(in.channels[i]);
					hipStreamDestroy(stream);
				}
			}
		}
			break;
		}

	}
} // namespace cu
} // namespace image
} // namespace wlib
#endif
