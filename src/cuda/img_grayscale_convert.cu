#include "hip/hip_runtime.h"
#include "../../include/cuda/weirdlib_cuda_image.hpp"
#include "./cuda_utils.cuh"
#ifdef WLIB_ENABLE_CUDA
#include <hip/hip_runtime.h>

namespace wlib
{
namespace image
{
namespace cu
{
	template<GrayscaleMethod method = GrayscaleMethod::Luminosity>
	__global__ void kernel_ConvertGrayscaleRGB(const float* __restrict__ r, const float* __restrict__ g, const float* __restrict__ b, float* __restrict__ outGray) {
		const int pixelID = getGlobalIdx_1x1();
		outGray[pixelID] = fminf(fmaf(r[pixelID], 0.2126f, fmaf(g[pixelID], 0.7152f, b[pixelID] * 0.0722f)), 255.0f);
	}

	template<>
	__global__ void kernel_ConvertGrayscaleRGB<GrayscaleMethod::Lightness>(const float* __restrict__ r, const float* __restrict__ g, const float* __restrict__ b, float* __restrict__ outGray) {
		const int pixelID = getGlobalIdx_1x1();
		outGray[pixelID] = ((fmaxf(fmaxf(r[pixelID], g[pixelID]), b[pixelID])) + (fminf(fminf(r[pixelID], g[pixelID]), b[pixelID]))) * 0.5f;
	}

	template<>
	__global__ void kernel_ConvertGrayscaleRGB<GrayscaleMethod::Average>(const float* __restrict__ r, const float* __restrict__ g, const float* __restrict__ b, float* __restrict__ outGray) {
		const int pixelID = getGlobalIdx_1x1();
		outGray[pixelID] = (r[pixelID] + g[pixelID] + b[pixelID]) * (1.0f/3.0f);
	}


	ImageSoACUDA& ConvertToGrayscale(ImageSoACUDA& inImg, const bool preserveAlpha, const GrayscaleMethod method) {
		if (inImg.format == F_GrayAlpha || inImg.format == F_Grayscale) {
			return inImg;
		}

		float* outGray;
		const size_t channelSize = inImg.width * inImg.height * sizeof(float);
		hipMalloc(&outGray, channelSize);

		float* red;
		float* green;
		float* blue;

		if (inImg.format == F_RGBA || inImg.format == F_RGB) {
			red = inImg.channels[0];
			green = inImg.channels[1];
			blue = inImg.channels[2];
		} else {
			red = inImg.channels[2];
			green = inImg.channels[1];
			blue = inImg.channels[0];
		}

		const size_t blockSize = getBlockSize(inImg.width * inImg.height);
		const size_t gridSize = inImg.width * inImg.height / blockSize;

		hipStream_t stream;
		hipStreamCreate(&stream);

		switch (method)
		{
		case GrayscaleMethod::Luminosity:
			kernel_ConvertGrayscaleRGB<GrayscaleMethod::Luminosity><<<gridSize, blockSize, 0, stream>>>(red, green, blue, outGray);
			break;
		case GrayscaleMethod::Lightness:
			kernel_ConvertGrayscaleRGB<GrayscaleMethod::Lightness><<<gridSize, blockSize, 0, stream>>>(red, green, blue, outGray);
			break;
		case GrayscaleMethod::Average:
			kernel_ConvertGrayscaleRGB<GrayscaleMethod::Average><<<gridSize, blockSize, 0, stream>>>(red, green, blue, outGray);
			break;
		}
		hipStreamDestroy(stream);

		if (!preserveAlpha || inImg.format == F_BGR || inImg.format == F_RGB) {
			inImg.format = F_Grayscale;
			for (auto& c : inImg.channels) {
				hipFree(c);
			}

			inImg.channels.resize(1);
			inImg.channels.shrink_to_fit();
			inImg.channels[0] = outGray;
		} else {
			inImg.format = F_GrayAlpha;
			for (size_t i = 0; i < 3; i++) {
				hipFree(inImg.channels[i]);
			}

			auto imgAlpha = inImg.channels[3];

			inImg.channels.resize(2);
			inImg.channels.shrink_to_fit();
			inImg.channels[0] = outGray;
			inImg.channels[1] = imgAlpha;
		}

		return inImg;
	}

} // namespace cu
} // namespace image
} // namespace wlib
#endif
